#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipcub/hipcub.hpp"
#include <vector>
#include <memory>
#include <string>
#include <exception>
#include <stdexcept>
#include <type_traits>
#include <thread>
#include <mutex>
#include "stdio.h"
#include <set>
#include <map>
#include <sstream>
#include <random>
#include <iostream>


#define M 8192
#define K 19456
#define N 14336
#define ITERATION 10 


#define DTOR_THROW noexcept(false) 


#define CUDA_CHECK(__cond)                              \
  do {                                                  \
    auto __err = (__cond);                              \
    if (__err != hipSuccess) {                         \
      auto __msg = hipGetErrorString(__err);           \
      throw std::runtime_error(std::string(__FILE__)    \
        + ":" + std::to_string(__LINE__) + ": "         \
        + #__cond + " failed with message : " + __msg   \
        + " , code : " + std::to_string(__err));        \
    }                                                   \
  } while (0)


#define CUBLAS_CHECK(__cond)                            \
  do {                                                  \
    auto __err = (__cond);                              \
    if (__err != HIPBLAS_STATUS_SUCCESS) {               \
      throw std::runtime_error(std::string(__FILE__)    \
        + ":" + std::to_string(__LINE__) + ": "         \
        + #__cond + " failed with code : "              \
        + std::to_string(__err));                       \
    }                                                   \
  } while (0)


#define DISABLE_COPY_AND_ASSIGN(classname)         \
 private:                                          \
  classname(const classname&) = delete;            \
  classname(classname&&) = delete;                 \
  classname& operator=(const classname&) = delete; \
  classname& operator=(classname&&) = delete
  

struct DeviceGuard {
  DISABLE_COPY_AND_ASSIGN(DeviceGuard);

 public:
  explicit DeviceGuard(int dev_id) {
    CUDA_CHECK(hipGetDevice(&dev_id_));
    CUDA_CHECK(hipSetDevice(dev_id));
  }

  ~DeviceGuard() DTOR_THROW {
    CUDA_CHECK(hipSetDevice(dev_id_));
  }
  
 private:
  int dev_id_;
}; 

struct CUDAResource {
  DISABLE_COPY_AND_ASSIGN(CUDAResource);

 public:
  int dev_id;
  hipStream_t stream;
  hipblasHandle_t handle;

  explicit CUDAResource(int dev_id) : dev_id(dev_id) {
    DeviceGuard guard(dev_id);
    CUDA_CHECK(hipStreamCreate(&stream));
    CUBLAS_CHECK(hipblasCreate(&handle)); 
    CUBLAS_CHECK(hipblasSetStream(handle, stream));
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  }

  ~CUDAResource() DTOR_THROW {
    SyncStream();
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void SyncStream() const {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
};


class Matrix {
  DISABLE_COPY_AND_ASSIGN(Matrix); 
  
 public:
  Matrix(const CUDAResource &resource, const float *cpu_data, int height, int width) {
    dev_id_ = resource.dev_id;
    height_ = height;
    width_ = width;

    DeviceGuard guard(dev_id_); 
    size_t nbytes = sizeof(data_[0]) * height_ * width_;
    resource.SyncStream();
    CUDA_CHECK(hipMalloc(&data_, nbytes));
    if (cpu_data != nullptr) {
      CUDA_CHECK(hipMemcpyAsync(data_, cpu_data, nbytes, hipMemcpyHostToDevice, resource.stream)); 
    }
    resource.SyncStream();
  }

  ~Matrix() DTOR_THROW {
    DeviceGuard guard(dev_id_);
    CUDA_CHECK(hipFree(data_));
  }

  void Matmul(const CUDAResource &resource, const Matrix &other, Matrix *z) const {
    DeviceGuard guard(dev_id_);
    if (width_ != other.height_) {
      throw std::runtime_error("Invalid Argument: " + std::to_string(width_) + " vs " + std::to_string(other.height_));
    }

    if (height_ != z->height_) {
      throw std::runtime_error("Invalid Argument: " + std::to_string(height_) + " vs " + std::to_string(z->height_));
    }

    if (other.width_ != z->width_) {
      throw std::runtime_error("Invalid Argument: " + std::to_string(other.width_) + " vs " + std::to_string(z->width_));
    }

    const auto *x_data = data_;
    const auto *y_data = other.data_; 
    int m = height_; 
    int k = width_;
    int n = other.width_; 
  
    int lda = k;
    int ldb = n;
    int ldc = n;

    using DType = typename std::remove_pointer<decltype(x_data)>::type;
    DType alpha = 1, beta = 0;
    CUBLAS_CHECK(cublasSgemmEx(
        resource.handle, 
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n,
        m,
        k,
        &alpha,
        y_data,
        HIP_R_32F,
        ldb,
        x_data,
        HIP_R_32F,
        lda,
        &beta,
        z->data_,
        HIP_R_32F,
        ldc));

    resource.SyncStream();
  } 

  std::vector<float> ToCPU(const CUDAResource &resource) const {
    DeviceGuard guard(dev_id_);
    size_t nbytes = height_ * width_ * sizeof(data_[0]);
    std::vector<float> out(height_ * width_);
    resource.SyncStream();
    CUDA_CHECK(hipMemcpyAsync(out.data(), data_, nbytes, hipMemcpyDeviceToHost, resource.stream)); 
    return out;
  }

 private:
  float *data_;
  int height_;
  int width_;
  int dev_id_;  
}; 


template <typename T>
bool IsEqual(const std::vector<T> &x, const std::vector<T> &y) {
  if (x.size() != y.size()) return false;
  bool is_equal = (std::memcmp(x.data(), y.data(), sizeof(T) * x.size()) == 0);
  return is_equal;
}


void ThreadMain(std::reference_wrapper<const CUDAResource> resource, const float *x_data, const float *y_data, 
                int m, int k, int n, int iteration, std::vector<float> *out, int *has_aadiff) {
  *has_aadiff = 0;
  auto &res = resource.get();
  Matrix x(res, x_data, m, k);
  Matrix y(res, y_data, k, n); 
  Matrix z(res, nullptr, m, n);

  auto compute = [&] {
    x.Matmul(res, y, &z); 
    return z.ToCPU(res);
  };

  *out = compute(); 
  for (int i = 1; i < iteration; ++i) {
    auto tmp_out = compute();
    if (!IsEqual(*out, tmp_out)) {
      *has_aadiff = 1;
    }
  }
}


template <typename T>
std::vector<std::vector<int>> FindEqualGroup(const std::vector<std::vector<T>> &data) {
  int n = static_cast<int>(data.size());
  if (n == 0) return {}; 

  std::map<int, std::set<int>> result;
  std::set<int> left;
  for (int i = 0; i < n; ++i) {
    left.insert(i);
  }

  while (!left.empty()) { 
    auto beg = left.begin();
    auto first_value = *beg;
    result[first_value].insert(first_value); 
    left.erase(beg);

    for (auto iter = left.begin(); iter != left.end(); ) {
      bool is_equal = IsEqual(data[first_value], data[*iter]);
      if (is_equal) {
        result[first_value].insert(*iter); 
        iter = left.erase(iter);
      }
    } 
  } 
  std::vector<std::vector<int>> group;
  for (const auto &pair : result) {
    group.emplace_back();
    group.back().assign(pair.second.begin(), pair.second.end());
  } 
  return group;
} 


template <typename T>
std::string VectorToString(const std::vector<T> &data) {
  std::stringstream ss;
  ss << "[";
  for (size_t i = 0; i < data.size(); ++i) {
    if (i > 0) {
      ss << ", "; 
    }
    ss << data[i];
  }
  ss << "]";
  return ss.str();
}


static unsigned int GetSeed() {
  std::random_device rd;
  return rd();
}

template <typename T, typename Generator>  
void GenerateData(std::vector<T> *data, Generator generator) {
  for (auto iter = data->begin(); iter != data->end(); ++iter) {
    *iter = static_cast<T>(generator());
  }
}

template <typename T>
std::vector<T> CPUMatmul(const T *x, const T *y, int m, int k, int n) {
  std::vector<T> z(m * n, static_cast<T>(0));
  for (int z_i = 0; z_i < m; ++z_i) {
    for (int z_j = 0; z_j < n; ++z_j) {
      auto &z_data = z[z_i * n + z_j]; 
      for (int x_k = 0; x_k < k; ++x_k) {
         auto &x_data = x[z_i * k + x_k];
         auto &y_data = y[x_k * n + z_j]; 
         z_data += (x_data * y_data);
      }
    }
  } 
  return z;
}

void TestMain() {
  int dev_cnt = -1;
  int rt_ver = -1, driver_ver = -1;
  CUDA_CHECK(hipGetDeviceCount(&dev_cnt));
  CUDA_CHECK(hipRuntimeGetVersion(&rt_ver));
  CUDA_CHECK(hipDriverGetVersion(&driver_ver));
  printf("Device Number : %d , Runtime Version : %d , Driver Version : %d\n", dev_cnt, rt_ver, driver_ver);

  CUDA_CHECK(hipSetDevice(0));
  std::vector<std::unique_ptr<CUDAResource>> resources;
  std::vector<std::vector<float>> outputs(dev_cnt);
  for (int i = 0; i < dev_cnt; ++i) {
    resources.emplace_back(new CUDAResource(i));
  }

  int m = M;
  int k = K;
  int n = N;
  int iteration = ITERATION; 

  std::vector<float> x(m * k);
  std::vector<float> y(k * n);

  auto seed = GetSeed();
  std::default_random_engine engine(seed);
  
  std::normal_distribution<float> dist(0.0, 0.01); 

  auto generator = [&engine, &dist] { return dist(engine); };
  GenerateData(&x, generator);
  GenerateData(&y, generator);

  std::vector<std::thread> threads; 
  std::vector<int> has_aadiff(dev_cnt, 0);
  for (int i = 0; i < dev_cnt; ++i) {
    auto &resource = *(resources[i]);
    threads.emplace_back(ThreadMain, std::cref(resource), x.data(), y.data(), m, k, n, iteration, &(outputs[i]), &(has_aadiff[i]));
  }

  for (auto &th : threads) {
    th.join();
  }

  std::vector<int> aadiff_devs;  
  for (int i = 0; i < dev_cnt; ++i) {
    if (has_aadiff[i]) {
      aadiff_devs.push_back(i);
    }
  }
  
  auto group = FindEqualGroup(outputs);
  std::string group_str;
  size_t i = 0;
  for (const auto &g : group) {
    if (i + 1 != group.size()) {
      group_str += " | ";
    } 
    group_str += "Group " + std::to_string(i) + " : ";
    group_str += VectorToString(g); 
    ++i;
  } 
  if (group.size() <= 1 && aadiff_devs.empty()) {
    group_str = "NoAADiff : " + group_str;
  } else {
    group_str = "HasAADiff : " + VectorToString(aadiff_devs) + " || " + group_str;
  }
  printf("%s\n", group_str.c_str());
}

int main() {
  TestMain();
  return 0;
}
